#include "hip/hip_runtime.h"
/*
*      ChatGPT was used to help with CUDA debugging. 
*
*
*/

#include <stdio.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

const char* main = "main";
const char* data_init = "data_init";
const char* correctness_check = "correctness_check";
const char* comm = "comm";
const char* comm_large = "comm_large";
const char* comm_small = "comm_small";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* comp_small = "comp_small";
const char* data_validation = "data_validation";

__global__ void merge(float *a, float *b, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int start = tid * n;
    int end = start + n;
    int i = start, j = start + n/2, k = start;

    while (i < start + n/2 && j < end) {
        if (a[i] <= a[j]) {
            b[k] = a[i];
            i++;
        } else {
            b[k] = a[j];
            j++;
        }
        k++;
    }
    while (i < start + n/2) {
        b[k] = a[i];
        i++;
        k++;
    }
    while (j < end) {
        b[k] = a[j];
        j++;
        k++;
    }
    for (int i = start; i < end; i++) {
        a[i] = b[i];
    }
}

void parallelMergeSort(float *a, float *b, int n) {
    int *dev_a, *dev_b;
    hipMalloc((void**)&dev_a, n * sizeof(int));
    hipMalloc((void**)&dev_b, n * sizeof(int));

    CALIPER_MARK_BEGIN(comm_large);
    hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    CALIPER_MARK_END(comm_large);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    CALIPER_MARK_BEGIN(comp);
    CALIPER_MARK_BEGIN(large_comp);
    for (int i = 2; i <= n; i *= 2) {
        for (int j = 0; j < n; j += i) {
            CALIPER_MARK_BEGIN(small_comp);
            merge<<<numBlocks, blockSize>>>(dev_a, dev_b, i);
            hipDeviceSynchronize();
            CALIPER_MARK_END(small_comp);
        }
    }
    CALIPER_MARK_END(large_comp);
    CALIPER_MARK_END(comp);

    hipMemcpy(a, dev_a, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
}

float randomFloat(int a, int b)
{
    if (a > b)
        return randomFloat(b, a);
    if (a == b)
        return a;
 
    return (float)randomInt(a, b) + randomFloat();
}

int main(int argc, char** argv) {
    cali::ConfigManager mgr;
    mgr.start();

    CALI_MARK_BEGIN(main);

    CALI_MARK_BEGIN(data_init);
    std::string arg = argv[1];
    const int n = std::stoi(arg);
    float *a = new float[n];

    for(int i = 0; i < n; i++){
        a[i] = randomFloat(1, 1000);
    }
    CALI_MARK_END(data_init);

    float *b = (float*)malloc(n * sizeof(float));

    parallelMergeSort(a, b, n);

    CALI_MARK_BEGIN(data_validation);
    printf("Sorted array: \n");
    for (int i = 1; i <= n; i++) {
        if(i != n && a[i] < a[i - 1]){
            printf("Not sorted\n");
            break;
        }
        printf("%d ", a[i - 1]);
    }
    printf("\n");
    CALI_MARK_END(data_validation);

    free(b);

    CALI_MARK_END(main);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "MergeSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "MPI"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(float)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", n); // The number of elements in input dataset (1000)
    adiak::value("InputType", "Random"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_procs", 1); // The number of processors (MPI ranks)
    adiak::value("num_threads", 0); // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", numBlocks); // The number of CUDA blocks 
    adiak::value("group_num", 20); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "AI"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    mgr.stop();
    mgr.flush();
}